#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "include/tiff_handler.h"

#define BLOCK_SIZE 256

float *imageA_d, *result_d;
hipfftComplex *data_d;

__global__ void normalize(int n, float *data); 
int convolve(int x, int y, float *imageA, float *result);
void finish(char *s);

int main(){

	float *img_inA, *img_out;

	hipHostMalloc((void**)&img_inA, sizeof(float)* 512 * 512);
	hipHostMalloc((void**)&img_out, sizeof(float)* 512 * 512);

	readTIFF((char*)"boy.tif", img_inA);

	convolve(512, 512, img_inA, img_out);

	writeTiff((char*)"out.tif", img_out, 512, 512);

	hipHostFree(img_inA); hipHostFree(img_out);

	return 0;
}

int convolve(int x, int y, float *imageA, float *result){

	hipfftHandle planFwd, planInv;
	hipError_t error1, error2, error3;
	hipfftResult_t status;

	int tStart = clock();

	error1 = hipMalloc((void**)&imageA_d, sizeof(float)*x*y);
	error2 = hipMalloc((void**)&result_d, sizeof(float)*x*y);
	error3 = hipMalloc((void**)&data_d, sizeof(hipfftComplex)*x*y);
	if ((error1 != hipSuccess) ||
		(error2 != hipSuccess) ||
		(error3 != hipSuccess)){
		finish((char*)"Memory Allocation failed\n");
		return EXIT_FAILURE;
	}
	
	error1 = hipMemcpy(imageA_d, imageA, sizeof(float)*x*y, hipMemcpyHostToDevice);
	if ((error1 != hipSuccess)){
		finish((char*)"Memcpy host to device failed\n");
		return EXIT_FAILURE;
	}

	//transform image into frequency domain
	status = hipfftPlan2d(&planFwd, x, y, HIPFFT_R2C);
	if (status != HIPFFT_SUCCESS){
		finish((char*)"creation of fft-plan failed\n");
		return EXIT_FAILURE;
	}

	status = hipfftExecR2C(planFwd, imageA_d, data_d);
	if (status != HIPFFT_SUCCESS){
		finish((char*)"hipfftExecR2C failed\n");
		return EXIT_FAILURE;
	}
	
	//transform image back
	status = hipfftPlan2d(&planInv, x, y, HIPFFT_C2R);
	if (status != HIPFFT_SUCCESS){
		finish((char*)"creation of fft-plan failed\n");
		return EXIT_FAILURE;
	}

	status = hipfftExecC2R(planInv, data_d, result_d);
	if (status != HIPFFT_SUCCESS){
		finish((char*)"hipfftExecR2C failed\n");
		return EXIT_FAILURE;
	}
	normalize << <(int)ceil(x*y/(float)BLOCK_SIZE), BLOCK_SIZE >> >(x*y, result_d);
	
	error1 = hipMemcpy(result, result_d, sizeof(float)*x*y, hipMemcpyDeviceToHost);
	if (error1 != hipSuccess){
		finish((char*)"(memcopy device to host failed\n");
		return EXIT_FAILURE;
	}	

	printf("Zeitdauer: %f\n", (clock() - tStart) / (float)CLOCKS_PER_SEC);

	return EXIT_SUCCESS;
}

void finish(char *s){
	if (s) printf(s);
	if (result_d)   hipFree(result_d);
	if (imageA_d)   hipFree(imageA_d);
	if (data_d)     hipFree(data_d);
} 

__global__ void normalize(int n, float *data){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < n){
		data[i] = data[i] / n;
	}
}
